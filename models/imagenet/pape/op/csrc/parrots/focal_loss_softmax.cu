#include <parrots/extension.hpp>
#include "parrots_cuda_helper.hpp"
using phalf=float16;
#include "focal_loss_softmax_kernel.cuh"

int SoftmaxFocalLossForwardLaucher(
    const int N, const DArrayLite logits,
    const DArrayLite targets, const float weight_pos,
    const float gamma, const float alpha,
    const int num_classes, DArrayLite losses,
    DArrayLite priors, hipStream_t stream){

    const int kThreadsPerBlock = 1024;
    int output_size = N;
    hipError_t err;

    err = hipGetLastError();
    if(hipSuccess != err)
    {
        fprintf( stderr, "%s#%d: cudaCheckError() failed : %s\n", __FILE__,
                __LINE__, hipGetErrorString( err ) );
        exit( -1 );
    }
    PARROTS_DISPATCH_FLOATING_TYPES_AND_HALF(
        logits.elemType().prim(), ([&] {
        // Grab the input tensor
        const scalar_t * logits_flat = logits.ptr<scalar_t>();
        const int * targets_flat = targets.ptr<int>();

        scalar_t * losses_flat = losses.ptr<scalar_t>();
        scalar_t * priors_flat = priors.ptr<scalar_t>();
            SpatialSoftmaxKernel<scalar_t><<<(output_size / num_classes + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
      N, logits_flat, priors_flat, num_classes);

            SoftmaxFocalLossKernel<scalar_t><<<(output_size / num_classes + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
      N, priors_flat, targets_flat, losses_flat, weight_pos, gamma, alpha, num_classes);
        }));

    err = hipGetLastError();
    if(hipSuccess != err)
    {
        fprintf( stderr, "%s#%d: cudaCheckError() failed : %s\n", __FILE__,
                __LINE__, hipGetErrorString( err ) );
        exit( -1 );
    }

    return 1;
}


int SoftmaxFocalLossBackwardLaucher(
    const int N, const DArrayLite logits, const DArrayLite targets,
    DArrayLite dX_data, const float weight_pos,
    const float gamma, const float alpha, const int num_classes, 
    const DArrayLite priors, DArrayLite buff, hipStream_t stream){

    const int kThreadsPerBlock = 1024;
    int output_size = N;

    PARROTS_DISPATCH_FLOATING_TYPES_AND_HALF(
        logits.elemType().prim(), ([&] {
    // Grab the input tensor
        const scalar_t * logits_flat = logits.ptr<scalar_t>();
        const int * targets_flat = targets.ptr<int>();
        const scalar_t * priors_flat = priors.ptr<scalar_t>();

        scalar_t * dX_data_flat = dX_data.ptr<scalar_t>();
        scalar_t * buff_flat = buff.ptr<scalar_t>();
            SoftmaxFocalLossGradientWeightKernel<scalar_t><<<(output_size / num_classes + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
                N, priors_flat, targets_flat, buff_flat, weight_pos, gamma, alpha, num_classes);

            SoftmaxFocalLossGradientKernel<scalar_t><<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
                N, priors_flat, targets_flat, buff_flat, dX_data_flat, num_classes);
        }));

    hipError_t err = hipGetLastError();
    if(hipSuccess != err)
    {
        fprintf( stderr, "%s#%d: cudaCheckError() failed : %s\n", __FILE__,
                __LINE__, hipGetErrorString( err ) );
        exit( -1 );
    }

    return 1;
}


