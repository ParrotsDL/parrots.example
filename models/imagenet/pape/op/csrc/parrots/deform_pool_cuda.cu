#include <parrots/extension.hpp>
#include "parrots_cuda_helper.hpp"
using phalf=float16;
#include "deform_pool_cuda_kernel.cuh"

void DeformablePSROIPoolForward(const DArrayLite data,
                                const DArrayLite bbox,
                                const DArrayLite trans,
                                DArrayLite out,
                                DArrayLite top_count,
                                const int batch,
                                const int channels,
                                const int height,
                                const int width,
                                const int num_bbox,
                                const int channels_trans,
                                const int no_trans,
                                const float spatial_scale,
                                const int output_dim,
                                const int group_size,
                                const int pooled_size,
                                const int part_size,
                                const int sample_per_part,
                                const float trans_std)
{
  const int pooled_height = pooled_size;
  const int pooled_width = pooled_size;
  const int count = num_bbox * output_dim * pooled_height * pooled_width;
  const int num_classes = no_trans ? 1 : channels_trans / 2;
  const int channels_each_class = no_trans ? output_dim : output_dim / num_classes;

  PARROTS_DISPATCH_FLOATING_TYPES_AND_HALF(
      data.elemType().prim(), ([&] {
        const scalar_t *bottom_data = data.ptr<scalar_t>();
        const scalar_t *bottom_rois = bbox.ptr<scalar_t>();
        const scalar_t *bottom_trans = no_trans ? NULL : trans.ptr<scalar_t>();
        scalar_t *top_data = out.ptr<scalar_t>();
        scalar_t *top_count_data = top_count.ptr<scalar_t>();

        DeformablePSROIPoolForwardKernel<<<GET_BLOCKS(count), CUDA_NUM_THREADS>>>(
            count, bottom_data, (scalar_t)spatial_scale, channels, height, width, pooled_height, pooled_width,
            bottom_rois, bottom_trans, no_trans, (scalar_t)trans_std, sample_per_part, output_dim,
            group_size, part_size, num_classes, channels_each_class, top_data, top_count_data);
      }));
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("error in DeformablePSROIPoolForward: %s\n", hipGetErrorString(err));
  }
}

void DeformablePSROIPoolBackwardAcc(const DArrayLite out_grad,
                                    const DArrayLite data,
                                    const DArrayLite bbox,
                                    const DArrayLite trans,
                                    const DArrayLite top_count,
                                    DArrayLite in_grad,
                                    DArrayLite trans_grad,
                                    const int batch,
                                    const int channels,
                                    const int height,
                                    const int width,
                                    const int num_bbox,
                                    const int channels_trans,
                                    const int no_trans,
                                    const float spatial_scale,
                                    const int output_dim,
                                    const int group_size,
                                    const int pooled_size,
                                    const int part_size,
                                    const int sample_per_part,
                                    const float trans_std)
{
  // LOG(INFO) << "DeformablePSROIPoolBackward";
  const int num_rois = num_bbox;
  const int pooled_height = pooled_size;
  const int pooled_width = pooled_size;
  const int count = num_bbox * output_dim * pooled_height * pooled_width;
  const int num_classes = no_trans ? 1 : channels_trans / 2;
  const int channels_each_class = no_trans ? output_dim : output_dim / num_classes;

  PARROTS_DISPATCH_FLOATING_TYPES_AND_HALF(
      out_grad.elemType().prim(), ([&] {
        const scalar_t *top_diff = out_grad.ptr<scalar_t>();
        const scalar_t *bottom_data = data.ptr<scalar_t>();
        const scalar_t *bottom_rois = bbox.ptr<scalar_t>();
        const scalar_t *bottom_trans = no_trans ? NULL : trans.ptr<scalar_t>();
        scalar_t *bottom_data_diff = in_grad.ptr<scalar_t>();
        scalar_t *bottom_trans_diff = no_trans ? NULL : trans_grad.ptr<scalar_t>();
        const scalar_t *top_count_data = top_count.ptr<scalar_t>();

        DeformablePSROIPoolBackwardAccKernel<<<GET_BLOCKS(count), CUDA_NUM_THREADS>>>(
            count, top_diff, top_count_data, num_rois, (scalar_t)spatial_scale, channels, height, width,
            pooled_height, pooled_width, output_dim, bottom_data_diff, bottom_trans_diff,
            bottom_data, bottom_rois, bottom_trans, no_trans, (scalar_t)trans_std, sample_per_part,
            group_size, part_size, num_classes, channels_each_class);
      }));
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("error in DeformablePSROIPoolForward: %s\n", hipGetErrorString(err));
  }
}
