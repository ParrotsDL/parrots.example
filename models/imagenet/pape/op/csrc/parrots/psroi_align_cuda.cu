#include <hip/hip_runtime.h>
#include <parrots/extension.hpp>
#include "parrots_cuda_helper.hpp"
#include "psroi_align_kernel.cuh"

int PSROIAlignForwardLauncher(
    DArrayLite bottom_data, const float spatial_scale,
    const int num_rois, const int output_dim,
    const int size_rois, const int height,
    const int width, const int channels,
    const int pooled_height, const int pooled_width,
    const float sampling_ratio, DArrayLite bottom_rois,
    DArrayLite top_data, DArrayLite mapping_channel)
{
    const int kThreadsPerBlock = 1024;
    int output_size = num_rois * pooled_height * pooled_width * output_dim;

    hipError_t err;
    err = hipGetLastError();
    
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s#%d: cudaCheckError() failed : %s\n", __FILE__, __LINE__,
            hipGetErrorString(err));
        exit(-1);
    }
    PARROTS_DISPATCH_FLOATING_TYPES_AND_HALF(
        bottom_data.elemType().prim(), ([&] {
            PSROIAlignForward<scalar_t>
                <<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock>>>(
                    output_size, bottom_data.ptr<scalar_t>(), spatial_scale,
                    channels, height, width, pooled_height, pooled_width,
                    bottom_rois.ptr<scalar_t>(), output_dim, pooled_height,
                    sampling_ratio, top_data.ptr<scalar_t>(),
                    mapping_channel.ptr<int>(), size_rois);
    }));

    err = hipGetLastError();
    
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s#%d: cudaCheckError() failed : %s\n", __FILE__, __LINE__,
            hipGetErrorString(err));
        exit(-1);
    }
    return 1;
}

int PSROIAlignBackwardLauncher(
    DArrayLite top_diff, const float spatial_scale,
    const int batch_size, const int num_rois,
    const int output_dim, const int size_rois,
    const int height, const int width,
    const int channels, const int pooled_height,
    const int pooled_width,
    const float sampling_ratio, DArrayLite bottom_rois,
    DArrayLite bottom_diff, DArrayLite mapping_channel)
{
    const int kThreadsPerBlock = 1024;
    // int output_size = batch_size * height * width * output_dim;
    int output_size = output_dim * pooled_height * pooled_width * num_rois;
    hipError_t err;

    PARROTS_DISPATCH_FLOATING_TYPES_AND_HALF(
        top_diff.elemType().prim(), ([&] {
            PSROIAlignBackward<scalar_t>
            <<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock>>>(
                output_size, top_diff.ptr<scalar_t>(),
                    mapping_channel.ptr<int>(), spatial_scale,
                    channels, height, width, pooled_height,
                    pooled_width, output_dim, pooled_height,
                    sampling_ratio, bottom_diff.ptr<scalar_t>(),
                    bottom_rois.ptr<scalar_t>(), size_rois);
    }));

    err = hipGetLastError();
    if (hipSuccess != err) 
    {
        fprintf(stderr, "%s#%d: cudaCheckError() failed : %s\n", __FILE__, __LINE__,
            hipGetErrorString(err));
        exit(-1);
    }

    return 1;
}
