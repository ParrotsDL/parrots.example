#include <ATen/ATen.h>
#include "pytorch_cuda_helper.hpp"
#include "psroi_mask_pooling_kernel.cuh"
using phalf=at::Tensor;
using at::Tensor;

int PSROIMaskPoolForwardLaucher(
    at::Tensor bottom_data,
    const float spatial_scale, const float roi_scale, const float bin_scale,
    const int num_rois, const int output_dim, const int size_rois,
    const int height, const int width, const int channels,
    const int pooled_height, const int pooled_width,
    at::Tensor bottom_rois, at::Tensor top_data, at::Tensor mapping_channel) {
    const int kThreadsPerBlock = 1024;
    int output_size = num_rois * pooled_height * pooled_width * output_dim;
    hipError_t err;

    err = hipGetLastError();
    if(hipSuccess != err) {
        fprintf( stderr, "%s#%d: cudaCheckError() failed : %s\n", __FILE__,
                __LINE__, hipGetErrorString( err ) );
        exit( -1 );
    }
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(bottom_data.type(),
                                        "psroi_mask_pooling_forward_cuda",
                                        ([&] {
        PSROIMaskPoolingForward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock>>>(
      output_size, bottom_data.data<scalar_t>(), spatial_scale, roi_scale, bin_scale, channels, height, width,
      pooled_height, pooled_width, bottom_rois.data<scalar_t>(), output_dim, pooled_height,
      top_data.data<scalar_t>(), mapping_channel.data<int>(), size_rois);
    }));
    // pooled_height == pooled_width == group_size
    err = hipGetLastError();
    if(hipSuccess != err) {
        fprintf( stderr, "%s#%d: cudaCheckError() failed : %s\n", __FILE__,
                __LINE__, hipGetErrorString( err ) );
        exit( -1 );
    }

    return 1;
}


int PSROIMaskPoolBackwardLaucher(
    at::Tensor top_diff, const float spatial_scale,
    const float roi_scale, const float bin_scale, const int batch_size, const int num_rois,
    const int output_dim, const int size_rois, const int height, const int width, const int channels,
    const int pooled_height, const int pooled_width,
    at::Tensor bottom_rois, at::Tensor bottom_diff, at::Tensor mapping_channel) {
    const int kThreadsPerBlock = 1024;
    //int output_size = batch_size * height * width * output_dim;
    int output_size = output_dim * pooled_height * pooled_width * num_rois;
    hipError_t err;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(top_diff.type(),
                                        "psroi_mask_pooling_backward_cuda",
                                        ([&] {
      PSROIMaskPoolingBackward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock>>>(
      output_size, top_diff.data<scalar_t>(), mapping_channel.data<int>(), num_rois,
      spatial_scale, roi_scale, bin_scale, channels,
      height, width, pooled_height, pooled_width, output_dim, 
      bottom_diff.data<scalar_t>(), bottom_rois.data<scalar_t>(), size_rois);
    }));

    err = hipGetLastError();
    if(hipSuccess != err) {
        fprintf( stderr, "%s#%d: cudaCheckError() failed : %s\n", __FILE__,
                __LINE__, hipGetErrorString( err ) );
        exit( -1 );
    }

    return 1;
}

